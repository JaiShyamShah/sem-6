#include "hip/hip_runtime.h"

#include <stdio.h>

__global__ void add_vectors(int *a, int *b, int *c, int n) {
    int i = blockDim.x * blockIdx.x + threadIdx.x; 
    if (i < n) {  // bounds checking to avoid accessing out of bounds
        c[i] = a[i] + b[i];
    }
}

int main(void) {
    int n = 256;  // Length of the vectors
    int *h_A = (int *)malloc(n * sizeof(int));
    int *h_B = (int *)malloc(n * sizeof(int));
    int *h_C = (int *)malloc(n * sizeof(int));

    // Initialize vectors A and B
    for (int i = 0; i < n; ++i) {
        h_A[i] = i;
        h_B[i] = i * 2;
    }

    int *d_A, *d_B, *d_C;

    // Allocate memory on device
    hipMalloc((void**)&d_A, n * sizeof(int));
    hipMalloc((void**)&d_B, n * sizeof(int));
    hipMalloc((void**)&d_C, n * sizeof(int));

    // Copy data from host to device
    hipMemcpy(d_A, h_A, n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, n * sizeof(int), hipMemcpyHostToDevice);

    // Launch kernel with 1 block and n threads
    add_vectors<<<1, n>>>(d_A, d_B, d_C, n);

    // Copy result back to host
    hipMemcpy(h_C, d_C, n * sizeof(int), hipMemcpyDeviceToHost);

    // Print the results
    printf("Results-1: ");
    for (int i = 0; i < n; ++i) {
        printf("%d ", h_C[i]);
    }
    printf("\n");

    // Launch kernel with n blocks and 1 thread per block
    add_vectors<<<n, 1>>>(d_A, d_B, d_C, n);
    hipMemcpy(h_C, d_C, n * sizeof(int), hipMemcpyDeviceToHost);

    // Print the results
    printf("Results-2: ");
    for (int i = 0; i < n; ++i) {
        printf("%d ", h_C[i]);
    }
    printf("\n");

    // Launch kernel with dynamic grid and block size
    dim3 dimGrid(ceil(n / 256.0), 1, 1);  // Dynamically calculate the number of blocks
    dim3 dimBlock(256, 1, 1);  // Block size of 256 threads
    add_vectors<<<dimGrid, dimBlock>>>(d_A, d_B, d_C, n);

    // Copy result back to host
    hipMemcpy(h_C, d_C, n * sizeof(int), hipMemcpyDeviceToHost);

    // Print the results
    printf("Results-3: ");
    for (int i = 0; i < n; ++i) {
        printf("%d ", h_C[i]);
    }
    printf("\n");

    // Free memory on the device and host
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(h_A);
    free(h_B);
    free(h_C);

    return 0;
}
